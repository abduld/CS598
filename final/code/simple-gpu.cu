
#include <hip/hip_runtime.h>
__device__ int ave(int a, int b)
{
   return (a+b)/2;
}

__global__ void simple(int *data)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   data[tid] = ave(tid, tid);
}


